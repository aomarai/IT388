#include "hip/hip_runtime.h"
/* File:     mat_add.cu
 * Purpose:  Implement matrix addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -arch=sm_21 -o mat_add mat_add.cu 
 * Run:      ./mat_add <m> <n>
 *              m is the number of rows
 *              n is the number of columns
 *
 * Input:    The matrices A and B
 * Output:   Result of matrix addition.  
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void addMatrix(float A[], float B[], float C[], int m, int n) 
{
   int ij = blockDim.x * blockIdx.x + threadIdx.x;

   if (blockIdx.x < m && threadIdx.x < n) 
      C[ij] = A[ij] + B[ij];
}

void Read_matrix(float A[], int m, int n) 
{
   int i, j;

   for (i = 0; i < m; i++)
      for (j = 0; j < n; j++)
         scanf("%f", &A[i*n+j]);
}

void Print_matrix(char title[], float A[], int m, int n) 
{
   int i, j;

   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*n+j]);
      printf("\n");
   }  
}

double getTime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec / 1000000.0; 
}


/* Host code */
int main(int argc, char* argv[]) {
   int m, n;
   float *h_A, *h_B, *h_C;
   float *d_A, *d_B, *d_C;
   size_t size;
   int blockSize, numBlocks;

   /* Get size of matrices */
   if (argc != 3) {
      fprintf(stderr, "usage: %s <row count> <col count>\n", argv[0]);
      exit(0);
   }
   m = strtol(argv[1], NULL, 10);
   n = m;
   printf("m = %d, n = %d\n", m, n);
   size = m*n*sizeof(float);

   h_A = (float*) malloc(size);
   h_B = (float*) malloc(size);
   h_C = (float*) malloc(size);
   
   printf("Enter the matrices A and B\n");
   Read_matrix(h_A, m, n);
   Read_matrix(h_B, m, n);

   Print_matrix("A =", h_A, m, n);
   Print_matrix("B =", h_B, m, n);

   /* Allocate matrices in device memory */
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, size);
   hipMalloc(&d_C, size);

   /* Copy matrices from host memory to device memory */
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   /* Invoke kernel using m thread blocks, each of    */
   /* which contains n threads                        */
   addMatrix<<<m, n>>>(d_A, d_B, d_C, m, n);

   /* Wait for the kernel to complete */
   hipDeviceSynchronize();

   /* Copy result from device memory to host memory */
   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   Print_matrix("The sum is: ", h_C, m, n);

   /* Free device memory */
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   /* Free host memory */
   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
}  /* main */