// Ashkan Omaraie IT388 Homework 5

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include <sys/time.h>
#include <math.h>

//Add matrix a and matrix b into matrix c
__global__ void addMatrix(int A[], int B[], int C[], int m, int n)
{
   int i= blockDim.x * blockIdx.x + threadIdx.x;

   if (blockIdx.x < m && threadIdx.x < n) 
      C[i] = A[i] + B[i];
}

//Fill the matrices with non-random numbers
void generateMatrices(int A[], int B[], int m, int n)
{
   int i, j;
   for (i = 0; i < m; i++)
   {
      for (j = 0; j < n; j++)
      {
         A[i * n + j] = i * j;
         B[i * n + j] = i * j;
      }
   }
}

void printMatrix(int A[], int m, int n)
{
   int i, j;
   for (i = 0; i < m; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d ", A[i * n + j]);
      printf("\n");
   }
}

// double getTime()
// {
//     struct timeval t;
//     gettimeofday(&t, NULL);
//     return t.tv_sec + t.tv_usec / 1000000.0;
// }

/* Host code */
int main(int argc, char *argv[])
{
   int m, n;
   int *h_A, *h_B, *h_C;
   int *d_A, *d_B, *d_C;
   size_t size;
   int blockSize, numBlocks;

   //Get size of matrices
   if (argc != 3)
   {
      fprintf(stderr, "usage: %s <matrix size> <threads per block>\n", argv[0]);
      exit(0);
   }
   m = n = atoi(argv[1]);
   blockSize = atoi(argv[2]);
   numBlocks = ceil(double(n) / blockSize);
   printf("Matrix Sizes: %d Block Size: %d\n", m, blockSize);
   size = m * n * sizeof(int);

   h_A = (int *)malloc(size);
   h_B = (int *)malloc(size);
   h_C = (int *)malloc(size);

   generateMatrices(h_A, h_B, m, n);

   // Allocate matrices in device memory
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, size);
   hipMalloc(&d_C, size);

   //Copy matrices from host memory to device memory
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   addMatrix<<<numBlocks, blockSize>>>(d_A, d_B, d_C, m, n);

   //Wait for the kernel to complete
   hipDeviceSynchronize();

   //Copy result from device memory to host memory
   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   printf("Sum:\n");
   printMatrix(h_C, m, n);

   //Free device memory
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   //Free host memory
   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
} /* main */